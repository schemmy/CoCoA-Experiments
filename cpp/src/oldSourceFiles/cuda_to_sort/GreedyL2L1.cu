//ulimit -s unlimited
//nvcc Cuda.cu  -arch sm_20 && ./a.out


#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "hip/device_functions.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

void checkCUDAError(const char *msg);


void main(){





}


void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(-1);
	}
}
