//ulimit -s unlimited
//nvcc -lcublas GreedyL2L1.cu

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "hip/device_functions.h"
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "hipblas.h"
#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

void modify(float *m, int ldm, int n, int p, int q, float alpha, float beta) {
	cublasSscal(n - p + 1, alpha, &m[IDX2F(p,q,ldm)], ldm);
	cublasSscal(ldm - p + 1, beta, &m[IDX2F(p,q,ldm)], 1);
}
#define M 6
#define N 5

void checkCUDAError(const char *msg);






int main() {
	int i, j;
	hipblasStatus_t stat;
	float* devPtrA;
	float* a = 0;
	a = (float *) malloc(M * N * sizeof(*a));
	if (!a) {
		printf("host memory allocation failed");
		return EXIT_FAILURE;
	}
	for (j = 1; j <= N; j++) {
		for (i = 1; i <= M; i++) {
			a[IDX2F(i,j,M)] = (i - 1) * M + j;
		}
	}
	cublasInit();
	stat = cublasAlloc(M * N, sizeof(*a), (void**) &devPtrA);

	if (stat != HIPBLAS_STATUS_SUCCESS) {
		printf("device memory allocation failed");
		return EXIT_FAILURE;
	}
	hipblasSetMatrix(M, N, sizeof(*a), a, M, devPtrA, M);

	for (j = 1; j <= N; j++) {
		for (i = 1; i <= M; i++) {
			printf("%7.0f", a[IDX2F(i,j,M)]);
		}
		printf("\n");
	}

//	modify(devPtrA, M, N, 2, 3, 16.0f, 12.0f);

	cublasSscal(M,10,&devPtrA[IDX2F(2,2,M)],2);


	hipblasGetMatrix(M, N, sizeof(*a), devPtrA, M, a, M);
	cublasFree(devPtrA);
	cublasShutdown();
	for (j = 1; j <= N; j++) {
		for (i = 1; i <= M; i++) {
			printf("%7.0f", a[IDX2F(i,j,M)]);
		}
		printf("\n");
	}
	return EXIT_SUCCESS;

}

void checkCUDAError(const char *msg) {
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
		exit(-1);
	}
}
